
#include <hip/hip_runtime.h>
#include <stdio.h>
/**********************
* using local memory *
**********************/
// a __device__
__global__ void use_local_memory_GPU(float in)

{
float f;
f = in;
// variable "f" is in local memory and private to each
// parameter "in" is in local memory and private to each// ... real code would presumably do other stuff here ...
}
/**********************
* using global memory *
**********************/
// a __global__ function runs on the GPU & can be called from host
__global__ void use_global_memory_GPU(float *array)
{
// "array" is a pointer into global memory on the device
array[threadIdx.x] = 2.0f * (float) threadIdx.x;
}
/**********************
* using shared memory *
**********************/
// (for clarity, hardcoding 128 threads/elements and omitting out-of-
__global__ void use_shared_memory_GPU(float *array)
{
// local variables, private to each thread
int i, index = threadIdx.x;
float average, sum = 0.0f;
// __shared__ variables are visible to all threads in the thread
// and have the same lifetime as the thread block
__shared__ float sh_arr[128];
// copy data from "array" in global memory to sh_arr in shared
// here, each thread is responsible for copying a single element.
sh_arr[index] = array[index];
__syncthreads();
// ensure all the writes to shared memory have
// now, sh_arr is fully populated. Let's find the average of all
for (i=0; i<index; i++) { sum += sh_arr[i]; }
average = sum / (index + 1.0f);
printf("Thread id = %d\t Average = %f\n",index,average);
// if array[index] is greater than the average of array[0..index-1],
// since array[] is in global memory, this change will be seen by the
// other thread blocks, if any)
if (array[index] > average) { array[index] = average; }
// the following code has NO EFFECT: it modifies shared memory, but
// the resulting modified data is never copied back to global memory// and vanishes when the thread block completes
sh_arr[index] = 3.14;
}
int main(int argc, char **argv)
{
/*
* First, call a kernel that shows using local memory
*/
use_local_memory_GPU<<<1, 128>>>(2.0f);
/*
* Next, call a kernel that shows using global memory
*/
float h_arr[128];
// convention: h_ variables live on host
float *d_arr;
// convention: d_ variables live on device (GPU
// allocate global memory on the device, place result in "d_arr"
hipMalloc((void **) &d_arr, sizeof(float) * 128);
// now copy data from host memory "h_arr" to device memory "d_arr"
hipMemcpy((void *)d_arr, (void *)h_arr, sizeof(float) * 128,hipMemcpyHostToDevice);
// launch the kernel (1 block of 128 threads)
use_global_memory_GPU<<<1, 128>>>(d_arr); // modifies the contents
// copy the modified array back to the host, overwriting contents ofh_arr
hipMemcpy((void *)h_arr, (void *)d_arr, sizeof(float) * 128,hipMemcpyDeviceToHost);
// ... do other stuff ...
/*
* Next, call a kernel that shows using shared memory
*/
// as before, pass in a pointer to data in global memory
use_shared_memory_GPU<<<1, 128>>>(d_arr);
// copy the modified array back to the host
hipMemcpy((void *)h_arr, (void *)d_arr, sizeof(float) * 128,hipMemcpyHostToDevice);
// ... do other stuff ...
// force the printf()s to flush
hipDeviceSynchronize();
return 0;
}
