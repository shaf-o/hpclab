
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
//Thread block size
#define BLOCK_SIZE 3
#define WA 3
// Matrix A width
#define HA 3
// Matrix A height
#define WB 3// Matrix B
#define HB WA
// Matrix B
#define WC WB
// Matrix C
#define HC HA
// Matrix C

//Allocates a matrix with random float entries.
void randomInit(float * data ,int size)
{
for(int i = 0; i < size; ++i)
//data[i] = rand() / (float) RAND_MAX;
data[i] = i;
}
// CUDA Kernel
__global__ void matrixMul(float* C,float* A,float* B,int wA,int wB)
{
// 2D Thread ID
int tx = threadIdx.x;
int ty = threadIdx.y;
// value stores the element that is computed by the thread
float value = 0;
for(int i = 0; i < wA; ++i)
{
float elementA = A[ty * wA + i];
float elementB = B[i * wB + tx];
value += elementA * elementB;
}
// Write the matrix to device memory each
// thread writes one element
C[ty * wA + tx] = value;
}
// Program main
int main(int argc ,char** argv)
{
// set seed for rand()
srand(2006);
// 1. allocate host memory for matrices A and B
unsigned int size_A = WA * HA;
unsigned int mem_size_A =sizeof(float) * size_A;
float* h_A = (float*) malloc(mem_size_A);
unsigned int size_B = WB * HB;
unsigned int mem_size_B =sizeof(float) * size_B;
float * h_B = (float*) malloc(mem_size_B);
// 2. initialize host memory
randomInit(h_A, size_A);
randomInit(h_B, size_B);// 3. print out A and B
printf("\n\nMatrix A\n");
for(int i = 0; i < size_A; i++)
{
printf("%f ", h_A[i]);
if(((i + 1) % WA) == 0)
printf("\n");
}
printf("\n\nMatrix B\n");
for(int i = 0; i < size_B; i++)
{
printf
("%f ", h_B[i]);
if(((i + 1) % WB) == 0)
printf("\n");
}
// 4. allocate host memory for the result C
unsigned int size_C = WC * HC;
unsigned int mem_size_C =sizeof(float) * size_C;
float * h_C = (float *) malloc(mem_size_C);
// 8. allocate device memory
float* d_A;
float* d_B;
hipMalloc((void**) &d_A, mem_size_A);
hipMalloc((void**) &d_B, mem_size_B);
//9. copy host memory to device
hipMemcpy(d_A, h_A,mem_size_A ,hipMemcpyHostToDevice);
hipMemcpy(d_B, h_B,mem_size_B ,hipMemcpyHostToDevice);
// 10. allocate device memory for the result
float* d_C;
hipMalloc((void**) &d_C, mem_size_C);
// 5. perform the calculation
//
//setup execution parameters
dim3 threads(BLOCK_SIZE , BLOCK_SIZE);
dim3 grid(WC / threads.x, HC / threads.y);
//
//execute the kernel
matrixMul<<< grid , threads >>>(d_C, d_A,d_B, WA, WB);
// 11. copy result from device to host
hipMemcpy(h_C, d_C, mem_size_C ,hipMemcpyDeviceToHost);
// 6. print out the results
printf("\n\n Matrix C ( Results ) \n ");
for(int i = 0;i<size_C; i ++){
printf("%f",h_C[i]);
if(((i+ 1) % WC) == 0)
printf("\n");
}printf("\n");
// 7.clean up memory
hipFree(d_A);
hipFree(d_B);
hipFree(d_C);
free(h_A);
free(h_B);
free(h_C);
}

